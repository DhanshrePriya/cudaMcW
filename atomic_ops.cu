
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void atomic_shared_kernel(int *output) {
    __shared__ int sharedVar;  
    if (threadIdx.x == 0) 
        sharedVar = 0;  

    
    sharedVar += 1; // op = 1

    //atomicAdd(&sharedVar, 1);  // op = 10

    __syncthreads();  

    if (threadIdx.x == 0) 
        *output = sharedVar;  
}

int main() {
    int h_result = 0, *d_result;
    hipMalloc(&d_result, sizeof(int));
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    atomic_shared_kernel<<<1, 10>>>(d_result);  

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    printf("Final shared memory value: %d\n", h_result);  

    hipFree(d_result);
    return 0;
}

