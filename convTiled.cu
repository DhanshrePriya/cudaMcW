#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define KERNEL_SIZE 3  
#define TILE_SIZE 16
__global__ void Conv2D(float* input, float* kernel, float* output, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int half_k = KERNEL_SIZE / 2;
    
    __shared__ float k[KERNEL_SIZE][KERNEL_SIZE]; 
    __shared__ float in[TILE_SIZE + 2][TILE_SIZE + 2]; 

    if (threadIdx.y < KERNEL_SIZE && threadIdx.x < KERNEL_SIZE) {
        k[threadIdx.y][threadIdx.x] = kernel[threadIdx.y * KERNEL_SIZE + threadIdx.x];
    }

    if (row < height && col < width) {
        in[threadIdx.y + 1][threadIdx.x + 1] = input[row * width + col];
    }

    __syncthreads();  

    if (row < half_k || row >= height - half_k || col < half_k || col >= width - half_k) {
        output[row * width + col] = input[row * width + col];
    }

    float sum = 0.0f;
    for (int i = -half_k; i <= half_k; i++) {
        for (int j = -half_k; j <= half_k; j++) {
            sum += in[threadIdx.y + 1 + i][threadIdx.x + 1 + j] * k[i + half_k][j + half_k];
        }
    }

    output[row * width + col] = sum;
}

void ConvolutionGPU(int width, int height, float* input, float* kernel, float* output) {
    float *d_input, *d_kernel, *d_output;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);

    hipMalloc(&d_input, width * height * sizeof(float));
    hipMalloc(&d_kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(float));
    hipMalloc(&d_output, width * height * sizeof(float));

    hipMemcpy(d_input, input, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + 15) / 16, (height + 15) / 16);

    Conv2D<<<gridSize, blockSize>>>(d_input, d_kernel, d_output, width, height);

    hipMemcpy(output, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, end);
    std::cout << "TIME: " << gpu_time << " ms" << std::endl;

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
}

int main() {
    int width = 1024, height = 1024;
    std::vector<float> input(width * height, 1.234f);
    std::vector<float> kernel = { 0, -1, 0, -1, 5, -1, 0, -1, 0 };  
    std::vector<float> output(width * height, 0.0f);

    ConvolutionGPU(width, height, input.data(), kernel.data(), output.data());

    return 0;
}
