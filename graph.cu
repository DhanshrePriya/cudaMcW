#include <hip/hip_runtime.h>

#include <iostream>

__global__ void incrementKernel(int *data, int increment) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    data[idx] += increment;
}

int main() {
    int *d_data;
    int h_data[1024] = {0}; 
    hipMalloc(&d_data, 1024 * sizeof(int));
    hipMemcpy(d_data, h_data, 1024 * sizeof(int), hipMemcpyHostToDevice);

    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Step 1: Create CUDA Graph
    hipGraphCreate(&graph, 0);

    // Step 2: Add Kernel Node to the Graph
    hipGraphNode_t kernelNode;
    int increment = 1;  // Initial increment
    void *kernelArgs[] = { &d_data, &increment };
    hipKernelNodeParams kernelParams = {0};

    kernelParams.func = (void*)incrementKernel;
    kernelParams.gridDim = dim3(1);
    kernelParams.blockDim = dim3(1024);
    kernelParams.sharedMemBytes = 0; 
    kernelParams.kernelParams = kernelArgs;
    
    hipGraphAddKernelNode(&kernelNode, graph, nullptr, 0, &kernelParams);

    // Step 3: Instantiate the Graph
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    // Step 4: Execute and Dynamically Update the Graph
    for (int i = 1; i <= 5; i++) {  
        increment = i;  // Update the increment value

        // Modify the kernel parameters with the new increment
        hipKernelNodeParams newParams = kernelParams;
        newParams.kernelParams = kernelArgs;  // Update kernel arguments

        // Apply the update to the existing graph execution instance
        hipGraphExecKernelNodeSetParams(graphExec, kernelNode, &newParams);

        // Launch the updated graph
        hipGraphLaunch(graphExec, stream);
        hipStreamSynchronize(stream);

        // Copy data back to host and print first 10 elements
        hipMemcpy(h_data, d_data, 1024 * sizeof(int), hipMemcpyDeviceToHost);
        std::cout << "Iteration " << i << ": ";
        for (int j = 0; j < 10; j++) {
            std::cout << h_data[j] << " ";
        }
        std::cout << std::endl;
    }

    // Cleanup
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    hipFree(d_data);
    hipStreamDestroy(stream);

    return 0;
}
