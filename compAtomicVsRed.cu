#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>

using namespace std;

#define N 1048576  
#define THREADS_PER_BLOCK 1024

__global__ void sum_atomic(float* d_in, float* d_out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        atomicAdd(d_out, d_in[idx]);
    }
}

__global__ void sum_reduction(float* d_in, float* d_out) {
    __shared__ float sdata[THREADS_PER_BLOCK];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    sdata[tid] = (idx < N) ? d_in[idx] : 0.0f;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(d_out, sdata[0]);
    }
}

int main() {
    vector<float> h_in(N);
    float *d_in, *d_out_atomic, *d_out_reduction;
    float sum_atomic_result = 0.0f, sum_reduction_result = 0.0f;

    srand(time(0));
    for (int i = 0; i < N; i++) {
        h_in[i] = static_cast<float>(rand() % 100 + 1); 
    }

    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out_atomic, sizeof(float));
    hipMalloc(&d_out_reduction, sizeof(float));

    hipMemcpy(d_in, h_in.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_out_atomic, 0, sizeof(float));
    hipMemset(d_out_reduction, 0, sizeof(float));

    hipEvent_t start, stop;
    float time_atomic, time_reduction;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sum_atomic<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_in, d_out_atomic);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_atomic, start, stop);

    hipEventRecord(start);
    sum_reduction<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_in, d_out_reduction);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_reduction, start, stop);

 
    hipMemcpy(&sum_atomic_result, d_out_atomic, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&sum_reduction_result, d_out_reduction, sizeof(float), hipMemcpyDeviceToHost);


    cout << "Sum (atomic): " << sum_atomic_result << " | Time: " << time_atomic << " ms\n";
    cout << "Sum (reduction): " << sum_reduction_result << " | Time: " << time_reduction << " ms\n";

    hipFree(d_in);
    hipFree(d_out_atomic);
    hipFree(d_out_reduction);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
