// naive_matmul_with_timing.cu
#include <iostream>
#include <hip/hip_runtime.h>

//#define N 512  // Size of the matrix (N x N)
#define EPSILON 1e-3  // For correctness check

// CUDA Kernel for naive matrix multiplication
__global__ void matmul_naive(const float* A, const float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Column index

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

// CPU reference matmul
void matmul_cpu(const float* A, const float* B, float* C, int n) {
    for (int row = 0; row < n; ++row) {
        for (int col = 0; col < n; ++col) {
            float sum = 0.0f;
            for (int k = 0; k < n; ++k) {
                sum += A[row * n + k] * B[k * n + col];
            }
            C[row * n + col] = sum;
        }
    }
}

// Function to check if two matrices are approximately equal
bool verify_result(const float* a, const float* b, int n) {
    for (int i = 0; i < n * n; ++i) {
        if (fabs(a[i] - b[i]) > EPSILON) {
            std::cout << "Mismatch at index " << i 
                      << ": GPU value = " << a[i]
                      << ", CPU value = " << b[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    int N = 1<<10;
    int size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C, *h_C_ref;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    h_C_ref = (float*)malloc(size);

    // Initialize input matrices
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;  // Or random values
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Create CUDA events
    hipEvent_t start_h2d, stop_h2d;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_d2h, stop_d2h;
    hipEventCreate(&start_h2d);
    hipEventCreate(&stop_h2d);
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_d2h);
    hipEventCreate(&stop_d2h);

    // Host to Device copy timing
    hipEventRecord(start_h2d);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(stop_h2d);

    // Kernel launch timing
    dim3 blockDim(32, 32);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y - 1) / blockDim.y);
    hipEventRecord(start_kernel);
    matmul_naive<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop_kernel);

    // Device to Host copy timing
    hipEventRecord(start_d2h);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop_d2h);

    // Wait for all events to complete
    hipEventSynchronize(stop_h2d);
    hipEventSynchronize(stop_kernel);
    hipEventSynchronize(stop_d2h);

    // Calculate elapsed times
    float time_h2d, time_kernel, time_d2h;
    hipEventElapsedTime(&time_h2d, start_h2d, stop_h2d);
    hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
    hipEventElapsedTime(&time_d2h, start_d2h, stop_d2h);

    std::cout << "H2D Copy Time (ms): " << time_h2d << std::endl;
    std::cout << "Kernel Execution Time (ms): " << time_kernel << std::endl;
    std::cout << "D2H Copy Time (ms): " << time_d2h << std::endl;

    // Compute CPU reference result
    matmul_cpu(h_A, h_B, h_C_ref, N);

    // Verify correctness
    if (verify_result(h_C, h_C_ref, N)) {
        std::cout << "Result is correct!" << std::endl;
    } else {
        std::cout << "Result is incorrect!" << std::endl;
    }

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);

    // Destroy events
    hipEventDestroy(start_h2d);
    hipEventDestroy(stop_h2d);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);
    hipEventDestroy(start_d2h);
    hipEventDestroy(stop_d2h);

    return 0;
}
