// tiled_matmul_with_timing.cu
#include <iostream>
#include <hip/hip_runtime.h>

#define N 512           // Size of the matrix (N x N)
#define TILE_SIZE 16    // Tile size (blockDim.x = blockDim.y)

#define EPSILON 1e-3    // For result verification

// CUDA Kernel: Matrix multiplication using shared memory tiling
__global__ void matmul_tiled(const float* A, const float* B, float* C, int n) {
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Loop over all tiles
    for (int t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load tile from global memory into shared memory
        if (row < n && (t * TILE_SIZE + threadIdx.x) < n) {
            tile_A[threadIdx.y][threadIdx.x] = A[row * n + t * TILE_SIZE + threadIdx.x];
        } else {
            tile_A[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < n && (t * TILE_SIZE + threadIdx.y) < n) {
            tile_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * n + col];
        } else {
            tile_B[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();  // Synchronize to make sure tiles are loaded

        // Multiply tiles
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];
        }

        __syncthreads();  // Synchronize before loading new tiles
    }

    if (row < n && col < n) {
        C[row * n + col] = sum;
    }
}

// CPU reference matmul
void matmul_cpu(const float* A, const float* B, float* C, int n) {
    for (int row = 0; row < n; ++row) {
        for (int col = 0; col < n; ++col) {
            float sum = 0.0f;
            for (int k = 0; k < n; ++k) {
                sum += A[row * n + k] * B[k * n + col];
            }
            C[row * n + col] = sum;
        }
    }
}

// Function to check if two matrices are approximately equal
bool verify_result(const float* a, const float* b, int n) {
    for (int i = 0; i < n * n; ++i) {
        if (fabs(a[i] - b[i]) > EPSILON) {
            std::cout << "Mismatch at index " << i 
                      << ": GPU value = " << a[i]
                      << ", CPU value = " << b[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    int size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C, *h_C_ref;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    h_C_ref = (float*)malloc(size);

    // Initialize input matrices
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;  // You can also use random values
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Create CUDA events
    hipEvent_t start_h2d, stop_h2d;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_d2h, stop_d2h;
    hipEventCreate(&start_h2d);
    hipEventCreate(&stop_h2d);
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_d2h);
    hipEventCreate(&stop_d2h);

    // Host to Device copy timing
    hipEventRecord(start_h2d);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(stop_h2d);

    // Kernel launch timing
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE,
                 (N + TILE_SIZE - 1) / TILE_SIZE);
    hipEventRecord(start_kernel);
    matmul_tiled<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop_kernel);

    // Device to Host copy timing
    hipEventRecord(start_d2h);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop_d2h);

    // Wait for all events to complete
    hipEventSynchronize(stop_h2d);
    hipEventSynchronize(stop_kernel);
    hipEventSynchronize(stop_d2h);

    // Calculate elapsed times
    float time_h2d, time_kernel, time_d2h;
    hipEventElapsedTime(&time_h2d, start_h2d, stop_h2d);
    hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
    hipEventElapsedTime(&time_d2h, start_d2h, stop_d2h);

    std::cout << "H2D Copy Time (ms): " << time_h2d << std::endl;
    std::cout << "Kernel Execution Time (ms): " << time_kernel << std::endl;
    std::cout << "D2H Copy Time (ms): " << time_d2h << std::endl;

    // Compute CPU reference result
    matmul_cpu(h_A, h_B, h_C_ref, N);

    // Verify correctness
    if (verify_result(h_C, h_C_ref, N)) {
        std::cout << "Result is correct!" << std::endl;
    } else {
        std::cout << "Result is incorrect!" << std::endl;
    }

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);

    // Destroy events
    hipEventDestroy(start_h2d);
    hipEventDestroy(stop_h2d);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);
    hipEventDestroy(start_d2h);
    hipEventDestroy(stop_d2h);

    return 0;
}
